#include "hip/hip_runtime.h"
#include <math.h>
#include <string>
#include <stdio.h>
#include "kernels_gpu.h"

const float      RISKFREE = 0.02f;
const float    VOLATILITY = 0.30f;

static double CND(double d)
{   
    const double       A1 = 0.31938153;
    const double       A2 = -0.356563782;
    const double       A3 = 1.781477937;
    const double       A4 = -1.821255978;
    const double       A5 = 1.330274429;
    const double RSQRT2PI = 0.39894228040143267793994605993438;
    
    double
    K = 1.0 / (1.0 + 0.2316419 * fabs(d));
    
    double
    cnd = RSQRT2PI * exp(- 0.5 * d * d) *
          (K * (A1 + K * (A2 + K * (A3 + K * (A4 + K * A5)))));
    
    if (d > 0)
        cnd = 1.0 - cnd;
    
    return cnd;
}

static void BlackScholesBodyCPU(
    float &callResult,
    float &putResult,
    float Sf, //Stock price
    float Xf, //Option strike
    float Tf, //Option years
    float Rf, //Riskless rate
    float Vf  //Volatility rate
)
{   
    double S = Sf, X = Xf, T = Tf, R = Rf, V = Vf;
    
    double sqrtT = sqrt(T);
    double    d1 = (log(S / X) + (R + 0.5 * V * V) * T) / (V * sqrtT);   
    double    d2 = d1 - V * sqrtT;
    double CNDD1 = CND(d1);
    double CNDD2 = CND(d2);
    
    //Calculate Call and Put simultaneously
    double expRT = exp(- R * T);
    callResult   = (float)(S * CNDD1 - X * expRT * CNDD2);
    putResult    = (float)(X * expRT * (1.0 - CNDD2) - S * (1.0 - CNDD1));
}

/*
    GPU blackscholes
    Reference: samples/
*/
void GpuKernel::blackscholes_2d(KernelParams& kernel_params, void** in_img, void** out_img){
}


#include <math.h>
#include <string>
#include <stdio.h>
#include "kernels_gpu.h"
#include "BmpUtil.h"
#include "dct8x8_kernel2.cuh"
#include "dct8x8_kernel_quantization.cuh"

#define BENCHMARK_SIZE 10
#define BLOCK_SIZE 8
#define BLOCK_SIZE2 64
#define BLOCK_SIZE_LOG2 3

//float C_a = 1.387039845322148f; //!< a = (2^0.5) * cos(    pi / 16);  Used in forward and inverse DCT.
//float C_b = 1.306562964876377f; //!< b = (2^0.5) * cos(    pi /  8);  Used in forward and inverse DCT.
//float C_c = 1.175875602419359f; //!< c = (2^0.5) * cos(3 * pi / 16);  Used in forward and inverse DCT.
//float C_d = 0.785694958387102f; //!< d = (2^0.5) * cos(5 * pi / 16);  Used in forward and inverse DCT.
//float C_e = 0.541196100146197f; //!< e = (2^0.5) * cos(3 * pi /  8);  Used in forward and inverse DCT.
//float C_f = 0.275899379282943f; //!< f = (2^0.5) * cos(7 * pi / 16);  Used in forward and inverse DCT.
//float C_norm = 0.3535533905932737f; // 1 / (8^0.5)

/*Already implemented in utils/BmpUtil.cpp*/
//float round_f(float num)
//{
//    float NumAbs = fabs(num);
//    int NumAbsI = (int)(NumAbs + 0.5f);
//    float sign = num > 0 ? 1.0f : -1.0f;
//    return sign * NumAbsI;
//}

/*
    GPU dct8x8
    Reference: samples/3_Imaging/dct8x8/dct8x8.cu: CUDA2
*/
void GpuKernel::dct8x8_2d(KernelParams& kernel_params, void** in_img, void** out_img){
    /* integration code */
    float* ImgF1   = reinterpret_cast<float*>(*in_img);
    float* out_tmp = reinterpret_cast<float*>(*out_img);
    // a hard-coded params that used by this kernel.
    //int ImgStride;

    //allocate device memory
    float *src, *dst;
    ROI Size;
    Size.width  = kernel_params.params.get_kernel_size();
    Size.height = kernel_params.params.get_kernel_size();
    /* integration code */
    int StrideF = (((int)ceil((Size.width*sizeof(float))/16.0f))*16) / sizeof(float);
 //   byte *ImgDst = MallocPlaneByte(Size.width, Size.height, &ImgStride);
    size_t DeviceStride;
    checkCudaErrors(hipMallocPitch((void **)&src, &DeviceStride, Size.width * sizeof(float), Size.height));
    checkCudaErrors(hipMallocPitch((void **)&dst, &DeviceStride, Size.width * sizeof(float), Size.height));
    DeviceStride /= sizeof(float);

    //copy from host memory to device
    checkCudaErrors(hipMemcpy2D(src, DeviceStride * sizeof(float),
                                 ImgF1, StrideF * sizeof(float),
                                 Size.width * sizeof(float), Size.height,
                                 hipMemcpyHostToDevice));

    dim3 GridFullWarps(Size.width / KER2_BLOCK_WIDTH, Size.height / KER2_BLOCK_HEIGHT, 1);
    dim3 ThreadsFullWarps(8, KER2_BLOCK_WIDTH/8, KER2_BLOCK_HEIGHT/8);

    //perform block-wise DCT processing and benchmarking
    const int numIterations = 100;

    for (int i = -1; i < numIterations; i++)
    {
        if (i == 0)
        {
            checkCudaErrors(hipDeviceSynchronize());
        }
 
        CUDAkernel2DCT<<<GridFullWarps, ThreadsFullWarps>>>(dst, src, (int)DeviceStride);
        getLastCudaError("Kernel execution failed");
    }
 
    checkCudaErrors(hipDeviceSynchronize());

    //setup execution parameters for quantization
    dim3 ThreadsSmallBlocks(BLOCK_SIZE, BLOCK_SIZE);
    dim3 GridSmallBlocks(Size.width / BLOCK_SIZE, Size.height / BLOCK_SIZE);

    // execute Quantization kernel
    CUDAkernelQuantizationFloat<<< GridSmallBlocks, ThreadsSmallBlocks >>>(dst, (int) DeviceStride);
    getLastCudaError("Kernel execution failed");
 
    //perform block-wise IDCT processing
    CUDAkernel2IDCT<<<GridFullWarps, ThreadsFullWarps >>>(src, dst, (int)DeviceStride);
    checkCudaErrors(hipDeviceSynchronize());
    getLastCudaError("Kernel execution failed");
    
    //copy quantized image block to host
    checkCudaErrors(hipMemcpy2D(out_tmp, StrideF *sizeof(float),
                                 src, DeviceStride *sizeof(float),
                                 Size.width *sizeof(float), Size.height,
                                 hipMemcpyDeviceToHost));
 
    //convert image back to byte representation
//    AddFloatPlane(128.0f, out_tmp, StrideF, Size);
//    CopyFloat2Byte(out_tmp, StrideF, ImgDst, ImgStride, Size);

    //clean up memory
    checkCudaErrors(hipFree(dst));
    checkCudaErrors(hipFree(src));
}


#include <string>
#include <stdio.h>
#include "cuda_utils.h"
#include "kernels_gpu.h"
#include "kernels_fft.cuh"
#include "kernels_fft_wrapper.cu"

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
//#include <hip/hip_runtime_api.h>
//#include <hip/hip_runtime.h>


void GpuKernel::fft_2d_input_conversion(){
    this->input_array_type.device_fp  = this->input_array_type.host_fp;
}

void GpuKernel::fft_2d_output_conversion(){
    Mat result;
    const int kernelH = 7;
    const int kernelW = 6;
    const int   dataH = kernel_params.params.get_kernel_size();
    const int   dataW = kernel_params.params.get_kernel_size();
    const int    fftH = snapTransformSize(dataH + kernelH - 1); 
    const int    fftW = snapTransformSize(dataW + kernelW - 1); 

    assert(this->output_array_type.device_fp != NULL);

    array2mat(result, this->output_array_type.device_fp, fftH, fftW);
    Mat cropped = result(Range(0, dataH), Range(0, dataW)); 
    mat2array(cropped, this->output_array_type.host_fp);
}    

/*
    GPU convolveFFT2D, this kernel used a fixed 7x6 convolving kernel.
    Reference: samples/3_Imaging/convolutionFFT2D/convolutionFFT2D.cu
*/
void GpuKernel::fft_2d(KernelParams& kernel_params, void** in_array, void** out_array){
    float* h_Data      = reinterpret_cast<float*>(*in_array);
    float* h_ResultGPU = reinterpret_cast<float*>(*out_array);
    
    float* h_Kernel;
    
    float* d_Data ;
    float* d_PaddedData;
    float* d_Kernel;
    float* d_PaddedKernel;

    fComplex* d_DataSpectrum;
    fComplex* d_KernelSpectrum;

    hipfftHandle fftPlanFwd, fftPlanInv;

    const int kernelH = 7;
    const int kernelW = 6;
    const int kernelY = 3;
    const int kernelX = 4;
    const int   dataH = kernel_params.params.get_kernel_size();
    const int   dataW = kernel_params.params.get_kernel_size();
    const int    fftH = snapTransformSize(dataH + kernelH - 1); 
    const int    fftW = snapTransformSize(dataW + kernelW - 1); 

    //printf("...allocating memory\n");
    float fft_2d_kernel_array[7*6] = {
        13, 12, 13,  0,  1,  1,
        0,  7,  8,  2,  8,  0,
        5,  9,  1, 11, 11,  3,
        14, 14,  8, 11,  0,  3,
        6,  8, 14, 13,  0, 10,
        10, 11, 14,  1,  2,  0,
        5, 15,  7,  5,  1,  7
    };
    h_Kernel = fft_2d_kernel_array;

    checkCudaErrors(hipMalloc((void **)&d_Data, dataH * dataW * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_Kernel, kernelH * kernelW * sizeof(float)));
    
    checkCudaErrors(hipMalloc((void **)&d_PaddedData,   fftH * fftW * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_PaddedKernel, fftH * fftW * sizeof(float)));

    checkCudaErrors(hipMalloc((void **)&d_DataSpectrum,   fftH * (fftW / 2 + 1) * sizeof(fComplex)));
    checkCudaErrors(hipMalloc((void **)&d_KernelSpectrum, fftH * (fftW / 2 + 1) * sizeof(fComplex)));
    checkCudaErrors(hipMemset(d_KernelSpectrum, 0, fftH * (fftW / 2 + 1) * sizeof(fComplex)));

    //printf("...creating R2C & C2R FFT plans for %i x %i\n", fftH, fftW);
    checkCudaErrors(hipfftPlan2d(&fftPlanFwd, fftH, fftW, HIPFFT_R2C));
    checkCudaErrors(hipfftPlan2d(&fftPlanInv, fftH, fftW, HIPFFT_C2R));

    //printf("...uploading to gpu and padding convolution kernel and input data\n");
    checkCudaErrors(hipMemcpy(d_Kernel, 
                               h_Kernel, 
                               kernelH * kernelW * sizeof(float), 
                               hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_Data,   
                               h_Data,   
                               dataH   * dataW *   sizeof(float), 
                               hipMemcpyHostToDevice));
    checkCudaErrors(hipMemset(d_PaddedKernel, 0, fftH * fftW * sizeof(float)));
    checkCudaErrors(hipMemset(d_PaddedData,   0, fftH * fftW * sizeof(float)));

    padKernel(
        d_PaddedKernel,
        d_Kernel,
        fftH,
        fftW,
        kernelH,
        kernelW,
        kernelY,
        kernelX
    );

    padDataClampToBorder(
        d_PaddedData,
        d_Data,
        fftH,
        fftW,
        dataH,
        dataW,
        kernelH,
        kernelW,
        kernelY,
        kernelX
    );

    //printf("...transforming convolution kernel\n");
    checkCudaErrors(hipfftExecR2C(fftPlanFwd, 
                 (hipfftReal *)d_PaddedKernel, 
                 (hipfftComplex *)d_KernelSpectrum));
    
    //printf("...running GPU FFT convolution\n");
    checkCudaErrors(hipDeviceSynchronize());
    
    checkCudaErrors(hipfftExecR2C(fftPlanFwd, 
                                 (hipfftReal *)d_PaddedData, 
                                 (hipfftComplex *)d_DataSpectrum));
    modulateAndNormalize(d_DataSpectrum, d_KernelSpectrum, fftH, fftW, 1);
    checkCudaErrors(hipfftExecC2R(fftPlanInv, 
                                 (hipfftComplex *)d_DataSpectrum, 
                                 (hipfftReal *)d_PaddedData));
 
    checkCudaErrors(hipDeviceSynchronize());
    
    float* tmp = (float *)malloc(fftH    * fftW * sizeof(float));;
    
    //printf("...reading back GPU convolution results\n");
    checkCudaErrors(hipMemcpy(tmp, d_PaddedData, fftH * fftW * sizeof(float), hipMemcpyDeviceToHost)); 
    h_ResultGPU = tmp;
    *out_array = (void*)h_ResultGPU;

    checkCudaErrors(hipfftDestroy(fftPlanInv));
    checkCudaErrors(hipfftDestroy(fftPlanFwd));

    checkCudaErrors(hipFree(d_DataSpectrum));
    checkCudaErrors(hipFree(d_KernelSpectrum));
    checkCudaErrors(hipFree(d_PaddedData));
    checkCudaErrors(hipFree(d_PaddedKernel));
    checkCudaErrors(hipFree(d_Data));
    checkCudaErrors(hipFree(d_Kernel));
}

#include <assert.h>
#include <iostream>
#include "cuda_utils.h"
#include "kernels_fft.cuh"

////////////////////////////////////////////////////////////////////////////////
/// Position convolution kernel center at (0, 0) in the image
////////////////////////////////////////////////////////////////////////////////
//extern "C" void padKernel(
//    float *d_Dst,
//    float *d_Src,
//    int fftH,
//    int fftW,
//    int kernelH,
//    int kernelW,
//    int kernelY,
//    int kernelX
//)
//{
//    assert(d_Src != d_Dst);
//    dim3 threads(32, 8);
//    dim3 grid(iDivUp(kernelW, threads.x), iDivUp(kernelH, threads.y));
// 
//    SET_FLOAT_BASE;
//#if (USE_TEXTURE)
//    hipTextureObject_t texFloat;
//    hipResourceDesc    texRes;
//    memset(&texRes,0,sizeof(hipResourceDesc));
// 
//    texRes.resType            = hipResourceTypeLinear;
//    texRes.res.linear.devPtr    = d_Src;
//    texRes.res.linear.sizeInBytes = sizeof(float)*kernelH*kernelW;
//    texRes.res.linear.desc = hipCreateChannelDesc<float>();
//
//    hipTextureDesc             texDescr;
//    memset(&texDescr,0,sizeof(hipTextureDesc));
//    texDescr.normalizedCoords = false;
//    texDescr.filterMode       = hipFilterModeLinear;
//    texDescr.addressMode[0] = hipAddressModeWrap;
//    texDescr.readMode = hipReadModeElementType;
//  
//    hipCreateTextureObject(&texFloat, &texRes, &texDescr, NULL);
//#endif
//  
//    padKernel_kernel<<<grid, threads>>>(
//        d_Dst,
//        d_Src,
//        fftH,
//        fftW,
//        kernelH,
//        kernelW,
//        kernelY,
//        kernelX
//#if (USE_TEXTURE)
//        , texFloat
//#endif
//    );
////    getLastCudaError("padKernel_kernel<<<>>> execution failed\n");
// 
//#if (USE_TEXTURE)
//    hipDestroyTextureObject(texFloat);
//#endif
//}

////////////////////////////////////////////////////////////////////////////////
// Prepare data for "pad to border" addressing mode
////////////////////////////////////////////////////////////////////////////////
//extern "C" void padDataClampToBorder(
//    float *d_Dst,
//    float *d_Src,
//    int fftH,
//    int fftW,
//    int dataH,
//    int dataW,
//    int kernelW,
//    int kernelH,
//    int kernelY,
//    int kernelX
//)
//{
//    assert(d_Src != d_Dst);
//    dim3 threads(32, 8);
//    dim3 grid(iDivUp(fftW, threads.x), iDivUp(fftH, threads.y));
//
//#if (USE_TEXTURE)
//    hipTextureObject_t texFloat;
//    hipResourceDesc            texRes;
//    memset(&texRes,0,sizeof(hipResourceDesc));
//
//    texRes.resType            = hipResourceTypeLinear;
//    texRes.res.linear.devPtr    = d_Src;
//    texRes.res.linear.sizeInBytes = sizeof(float)*dataH*dataW;
//    texRes.res.linear.desc = hipCreateChannelDesc<float>();
// 
//    hipTextureDesc             texDescr;
//    memset(&texDescr,0,sizeof(hipTextureDesc));
// 
//    texDescr.normalizedCoords = false;
//    texDescr.filterMode       = hipFilterModeLinear;
//    texDescr.addressMode[0] = hipAddressModeWrap;
//    texDescr.readMode = hipReadModeElementType;
// 
//    hipCreateTextureObject(&texFloat, &texRes, &texDescr, NULL);
//#endif
// 
//    padDataClampToBorder_kernel<<<grid, threads>>>(
//        d_Dst,
//        d_Src,
//        fftH,
//        fftW,
//        dataH,
//        dataW,
//        kernelH,
//        kernelW,
//        kernelY,
//        kernelX
//#if (USE_TEXTURE)
//       ,texFloat
//#endif
//    );
////    getLastCudaError("padDataClampToBorder_kernel<<<>>> execution fai    led\n");
// 
//#if (USE_TEXTURE)
//    hipDestroyTextureObject(texFloat);
//#endif
//}

////////////////////////////////////////////////////////////////////////////////
// Modulate Fourier image of padded data by Fourier image of padded kernel
// and normalize by FFT size
////////////////////////////////////////////////////////////////////////////////
//extern "C" void modulateAndNormalize(
//    fComplex *d_Dst,
//    fComplex *d_Src,
//    int fftH,
//    int fftW,
//    int padding
//)
//{
//    assert(fftW % 2 == 0);
//    const int dataSize = fftH * (fftW / 2 + padding);
//
//    modulateAndNormalize_kernel<<<iDivUp(dataSize, 256), 256>>>(
//        d_Dst,
//        d_Src,
//        dataSize,
//        1.0f / (float)(fftW *fftH)
//    );
////    getLastCudaError("modulateAndNormalize() execution failed\n");
//}


//void fft_2d_input_conversion_wrapper(){
//    return;
//}

#include "kernels_gpu.h"

#include <hip/hip_runtime.h>

#ifdef RD_WG_SIZE_0_0                                                            
        #define BLOCK_SIZE RD_WG_SIZE_0_0                                        
#elif defined(RD_WG_SIZE_0)                                                      
        #define BLOCK_SIZE RD_WG_SIZE_0                                          
#elif defined(RD_WG_SIZE)                                                        
        #define BLOCK_SIZE RD_WG_SIZE                                            
#else
        #define BLOCK_SIZE 16                                                            
#endif

/* some constants */
#define chip_height 0.016
#define chip_width 0.016
#define t_chip 0.0005
#define PRECISION 0.001
#define SPEC_HEAT_SI 1.75e6
#define K_SI 100
#define FACTOR_CHIP 0.5
#define MAX_PD 3.0e6

#define IN_RANGE(x, min, max)   ((x)>=(min) && (x)<=(max))
#define CLAMP_RANGE(x, min, max) x = (x<(min)) ? min : ((x>(max)) ? max : x )
//#define MIN(a, b) ((a)<=(b) ? (a) : (b))
 
__global__ void calculate_temp(int iteration,  //number of iteration
                               float *power,   //power input
                               float *temp_src,    //temperature input/output
                               float *temp_dst,    //temperature input/output
                               int grid_cols,  //Col of grid
                               int grid_rows,  //Row of grid
                               int border_cols,  // border offset 
                               int border_rows,  // border offset
                               float Cap,      //Capacitance
                               float Rx,
                               float Ry,
                               float Rz,
                               float step,
                               float time_elapsed){

        __shared__ float temp_on_cuda[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float power_on_cuda[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float temp_t[BLOCK_SIZE][BLOCK_SIZE]; // saving temparary temperature result
 
    float amb_temp = 80.0;
        float step_div_Cap;
        float Rx_1,Ry_1,Rz_1;

    int bx = blockIdx.x;
        int by = blockIdx.y;
 
    int tx=threadIdx.x;
    int ty=threadIdx.y;
 
    step_div_Cap=step/Cap;

    Rx_1=1/Rx;
    Ry_1=1/Ry;
    Rz_1=1/Rz;
 
        // each block finally computes result for a small block
        // after N iterations. 
        // it is the non-overlapping small blocks that cover 
        // all the input data
 
        // calculate the small block size
    int small_block_rows = BLOCK_SIZE-iteration*2;//EXPAND_RATE
    int small_block_cols = BLOCK_SIZE-iteration*2;//EXPAND_RATE

        // calculate the boundary for the block according to 
        // the boundary of its small block
        int blkY = small_block_rows*by-border_rows;
        int blkX = small_block_cols*bx-border_cols;
        int blkYmax = blkY+BLOCK_SIZE-1;
        int blkXmax = blkX+BLOCK_SIZE-1;

        // calculate the global thread coordination
    int yidx = blkY+ty;
    int xidx = blkX+tx;
 
        // load data if it is within the valid input range
    int loadYidx=yidx, loadXidx=xidx;
        int index = grid_cols*loadYidx+loadXidx;
 
    if(IN_RANGE(loadYidx, 0, grid_rows-1) && IN_RANGE(loadXidx, 0, grid_cols-1)){
            temp_on_cuda[ty][tx] = temp_src[index];  // Load the temperature data from global memory to shared memory
            power_on_cuda[ty][tx] = power[index];// Load the power data from global memory to shared memory
    }
    __syncthreads();

        // effective range within this block that falls within 
        // the valid range of the input data
        // used to rule out computation outside the boundary.
        int validYmin = (blkY < 0) ? -blkY : 0;
        int validYmax = (blkYmax > grid_rows-1) ? BLOCK_SIZE-1-(blkYmax-grid_rows+1) : BLOCK_SIZE-1;
        int validXmin = (blkX < 0) ? -blkX : 0;
        int validXmax = (blkXmax > grid_cols-1) ? BLOCK_SIZE-1-(blkXmax-grid_cols+1) : BLOCK_SIZE-1;
 
        int N = ty-1;
        int S = ty+1;
        int W = tx-1;
        int E = tx+1;
 
        N = (N < validYmin) ? validYmin : N;
        S = (S > validYmax) ? validYmax : S;
        W = (W < validXmin) ? validXmin : W;
        E = (E > validXmax) ? validXmax : E;
 
        bool computed;
        for (int i=0; i<iteration ; i++){
            computed = false;
            if( IN_RANGE(tx, i+1, BLOCK_SIZE-i-2) &&  \
                  IN_RANGE(ty, i+1, BLOCK_SIZE-i-2) &&  \
                  IN_RANGE(tx, validXmin, validXmax) && \
                  IN_RANGE(ty, validYmin, validYmax) ) {
                  computed = true;
                  temp_t[ty][tx] =   temp_on_cuda[ty][tx] + step_div_Cap * (power_on_cuda[ty][tx] +
                     (temp_on_cuda[S][tx] + temp_on_cuda[N][tx] - 2.0*temp_on_cuda[ty][tx]) * Ry_1 +
                     (temp_on_cuda[ty][E] + temp_on_cuda[ty][W] - 2.0*temp_on_cuda[ty][tx]) * Rx_1 +
                     (amb_temp - temp_on_cuda[ty][tx]) * Rz_1);
 
            }
            __syncthreads();
            if(i==iteration-1)
                break;
            if(computed)     //Assign the computation range
                temp_on_cuda[ty][tx]= temp_t[ty][tx];
            __syncthreads();
          }
 
      // update the global memory
      // after the last iteration, only threads coordinated within the 
      // small block perform the calculation and switch on ``computed''
      if (computed){
          temp_dst[index]= temp_t[ty][tx];
      }
}
/*
   compute N time steps
*/
 
int compute_tran_temp(float *MatrixPower,float *MatrixTemp[2], int col, int row, \
        int total_iterations, int num_iterations, int blockCols, int blockRows, int borderCols, int borderRows)
{
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
        dim3 dimGrid(blockCols, blockRows);
     
    float grid_height = chip_height / row;
    float grid_width = chip_width / col;
     
    float Cap = FACTOR_CHIP * SPEC_HEAT_SI * t_chip * grid_width * grid_height;
    float Rx = grid_width / (2.0 * K_SI * t_chip * grid_height);
    float Ry = grid_height / (2.0 * K_SI * t_chip * grid_width);
    float Rz = t_chip / (K_SI * grid_height * grid_width);
     
    float max_slope = MAX_PD / (FACTOR_CHIP * t_chip * SPEC_HEAT_SI);
    float step = PRECISION / max_slope;
    float t;
        float time_elapsed;
    time_elapsed=0.001;
 
        int src = 1, dst = 0;
 
    for (t = 0; t < total_iterations; t+=num_iterations) {
            int temp = src;
            src = dst;
            dst = temp;
            calculate_temp<<<dimGrid, dimBlock>>>(MIN(num_iterations, total_iterations-t), MatrixPower,MatrixTemp[src],MatrixTemp[dst],\
        col,row,borderCols, borderRows, Cap,Rx,Ry,Rz,step,time_elapsed);
    }
        return dst;
}

/* Reference code: rodinia_3.1/cuda/hotspot/hotspot.cu */
void GpuKernel::hotspot_2d(KernelParams& kernel_params, void** input, void** output){

    int dim = kernel_params.params.get_kernel_size();
    int grid_rows = dim;
    int grid_cols = dim;
    int size = dim * dim;

    /* some constants */
    int total_iterations = 1;
    int pyramid_height = 1;

    /* pyramid parameters */
    # define EXPAND_RATE 2// add one iteration will extend the pyramid base by 2 per each borderline
    int borderCols = (pyramid_height)*EXPAND_RATE/2;
    int borderRows = (pyramid_height)*EXPAND_RATE/2;
    int smallBlockCol = BLOCK_SIZE-(pyramid_height)*EXPAND_RATE;
    int smallBlockRow = BLOCK_SIZE-(pyramid_height)*EXPAND_RATE;
    int blockCols = grid_cols/smallBlockCol+((grid_cols%smallBlockCol==0)?0:1);
    int blockRows = grid_rows/smallBlockRow+((grid_rows%smallBlockRow==0)?0:1);

    /* host pointers */
    float* host_float_ptr = reinterpret_cast<float*>(*input);
    float* FilesavingTemp = host_float_ptr;
    float* FilesavingPower = &host_float_ptr[size];
    float* MatrixOut = reinterpret_cast<float*>(*output);

    /* device pointers */
    float *MatrixTemp[2], *MatrixPower;
    hipMalloc((void**)&MatrixTemp[0], sizeof(float)*size);
    hipMalloc((void**)&MatrixTemp[1], sizeof(float)*size);
    hipMemcpy(MatrixTemp[0], FilesavingTemp, sizeof(float)*size, hipMemcpyHostToDevice);
 
    hipMalloc((void**)&MatrixPower, sizeof(float)*size);
    hipMemcpy(MatrixPower, FilesavingPower, sizeof(float)*size, hipMemcpyHostToDevice);
    printf("Start computing the transient temperature\n");
    int ret = compute_tran_temp(MatrixPower,MatrixTemp,grid_cols,grid_rows, \
     total_iterations,pyramid_height, blockCols, blockRows, borderCols, borderRows);
    printf("Ending simulation\n");
    hipMemcpy(MatrixOut, MatrixTemp[ret], sizeof(float)*size, hipMemcpyDeviceToHost);
 
    //writeoutput(MatrixOut,grid_rows, grid_cols, ofile);
 
    hipFree(MatrixPower);
    hipFree(MatrixTemp[0]);
    hipFree(MatrixTemp[1]);
}
#include "kernels_gpu.h"
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/core/core.hpp>

void GpuKernel::kmeans_2d(const cuda::GpuMat in_img, cuda::GpuMat& out_img){
}
#include <string>
#include <stdio.h>
#include <opencv2/cudaarithm.hpp>
#include <opencv2/cudafilters.hpp>
#include "kernels_gpu.h"

void GpuKernel::laplacian_2d(const cuda::GpuMat in_img, cuda::GpuMat& out_img){
    auto laplacian = cuda::createLaplacianFilter(in_img.type(), in_img.type(), 3/*kernel size*/, 1/*scale*/, BORDER_DEFAULT);
    laplacian->apply(in_img, out_img);
    cuda::abs(out_img, out_img);
}
#include <string>
#include <stdio.h>
#include <opencv2/cudafilters.hpp> // create[XXX]Filter()
#include "kernels_gpu.h"

void GpuKernel::mean_2d(const cuda::GpuMat in_img, cuda::GpuMat& out_img){
    auto median = cuda::createBoxFilter(in_img.type(), in_img.type(), Size(3, 3),     Point(-1, -1), BORDER_DEFAULT);
    median->apply(in_img, out_img);
}
#include <string>
#include <stdio.h>
#include "kernels_gpu.h"

void GpuKernel::minimum_2d(const cuda::GpuMat in_img, cuda::GpuMat& out_img){
    out_img = in_img;
}
#include <string>
#include <stdio.h>
#include <opencv2/cudaarithm.hpp> // addWeighted()
#include <opencv2/cudafilters.hpp> // create[XXX]Filter()
#include "kernels_gpu.h"
void GpuKernel::sobel_2d(const cuda::GpuMat in_img, cuda::GpuMat& out_img){

    cuda::GpuMat grad_x, grad_y;
    cuda::GpuMat abs_grad_x, abs_grad_y;

    int ddepth = CV_32F;
    auto sobel_dx = cuda::createSobelFilter(in_img.type(), ddepth, 1, 0, 3);
    auto sobel_dy = cuda::createSobelFilter(in_img.type(), ddepth, 0, 1, 3);
 
    sobel_dx->apply(in_img, grad_x);
    sobel_dy->apply(in_img, grad_y);
 
    cuda::abs(grad_x, abs_grad_x);
    cuda::abs(grad_y, abs_grad_y);
  
    cuda::addWeighted(abs_grad_x, 0.5, abs_grad_y, 0.5, 0, out_img);
}
#include "srad.h"
#include "kernels_gpu.h"
#include "srad_kernel.cu"


void GpuKernel::srad_2d(KernelParams& kernel_params, void** input, void** output){
    int rows = kernel_params.params.get_kernel_size();
    int cols = kernel_params.params.get_kernel_size();
    int size_I, size_R, niter = 1, iter;
    float *I, *J, lambda, q0sqr, sum, sum2, tmp, meanROI, varROI;

    float *J_cuda;
    float *C_cuda;
    float *E_C, *W_C, *N_C, *S_C;

    unsigned int r1 = 0, r2 = rows-1, c1 = 0, c2 = cols-1; // need init
    float *c;

    size_I = cols * rows;
    size_R = (r1-r1+1)*(c2-c1+1);
 
    I = (float*)*input;
    J = (float*)*output;
    c = (float *)malloc(sizeof(float)* size_I);

    //Allocate device memory
    hipMalloc((void**)& J_cuda, sizeof(float)* size_I);
    hipMalloc((void**)& C_cuda, sizeof(float)* size_I);
    hipMalloc((void**)& E_C, sizeof(float)* size_I);
    hipMalloc((void**)& W_C, sizeof(float)* size_I);
    hipMalloc((void**)& S_C, sizeof(float)* size_I);
    hipMalloc((void**)& N_C, sizeof(float)* size_I);

    for (int k = 0;  k < size_I; k++ ) {
        J[k] = (float)exp(I[k]) ;
    }

    for(iter=0; iter < niter ; iter++){
        sum=0; sum2=0;
        for (int i=r1; i<=r2; i++) {
            for (int j=c1; j<=c2; j++) {
                tmp   = J[i * cols + j];
                sum  += tmp ;
                sum2 += tmp*tmp;
            }
        }
        meanROI = sum / size_R;
        varROI  = (sum2 / size_R) - meanROI*meanROI;
        q0sqr   = varROI / (meanROI*meanROI);

        //Currently the input size must be divided by 16 - the block size
        int block_x = cols/BLOCK_SIZE ;
        int block_y = rows/BLOCK_SIZE ;
 
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
        dim3 dimGrid(block_x , block_y);
 
        //Copy data from main memory to device memory
        hipMemcpy(J_cuda, J, sizeof(float) * size_I, hipMemcpyHostToDevice);
 
        //Run kernels
        srad_cuda_1<<<dimGrid, dimBlock>>>(E_C, W_C, N_C, S_C, J_cuda, C_cuda,     cols, rows, q0sqr);
        srad_cuda_2<<<dimGrid, dimBlock>>>(E_C, W_C, N_C, S_C, J_cuda, C_cuda,     cols, rows, lambda, q0sqr);
 
        //Copy data from device memory to main memory
        hipMemcpy(J, J_cuda, sizeof(float) * size_I, hipMemcpyDeviceToHost);
    }
    hipDeviceSynchronize();
    //hipDeviceSynchronize();

    hipFree(C_cuda);
    hipFree(J_cuda);
    hipFree(E_C);
    hipFree(W_C);
    hipFree(N_C);
    hipFree(S_C);
    free(c);
}
#include "srad.h"
#include <stdio.h>

//__global__ void
//srad_cuda_1(
//		  float *E_C, 
//		  float *W_C, 
//		  float *N_C, 
//		  float *S_C,
//		  float * J_cuda, 
//		  float * C_cuda, 
//		  int cols, 
//		  int rows, 
//		  float q0sqr
//) 
//{
//
//  //block id
//  int bx = blockIdx.x;
//  int by = blockIdx.y;
//
//  //thread id
//  int tx = threadIdx.x;
//  int ty = threadIdx.y;
//  
//  //indices
//  int index   = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + tx;
//  int index_n = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + tx - cols;
//  int index_s = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * BLOCK_SIZE + tx;
//  int index_w = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty - 1;
//  int index_e = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + BLOCK_SIZE;
//
//  float n, w, e, s, jc, g2, l, num, den, qsqr, c;
//
//  //shared memory allocation
//  __shared__ float temp[BLOCK_SIZE][BLOCK_SIZE];
//  __shared__ float temp_result[BLOCK_SIZE][BLOCK_SIZE];
//
//  __shared__ float north[BLOCK_SIZE][BLOCK_SIZE];
//  __shared__ float south[BLOCK_SIZE][BLOCK_SIZE];
//  __shared__ float  east[BLOCK_SIZE][BLOCK_SIZE];
//  __shared__ float  west[BLOCK_SIZE][BLOCK_SIZE];
//
//  //load data to shared memory
//  north[ty][tx] = J_cuda[index_n]; 
//  south[ty][tx] = J_cuda[index_s];
//  if ( by == 0 ){
//  north[ty][tx] = J_cuda[BLOCK_SIZE * bx + tx]; 
//  }
//  else if ( by == gridDim.y - 1 ){
//  south[ty][tx] = J_cuda[cols * BLOCK_SIZE * (gridDim.y - 1) + BLOCK_SIZE * bx + cols * ( BLOCK_SIZE - 1 ) + tx];
//  }
//   __syncthreads();
// 
//  west[ty][tx] = J_cuda[index_w];
//  east[ty][tx] = J_cuda[index_e];
//
//  if ( bx == 0 ){
//  west[ty][tx] = J_cuda[cols * BLOCK_SIZE * by + cols * ty]; 
//  }
//  else if ( bx == gridDim.x - 1 ){
//  east[ty][tx] = J_cuda[cols * BLOCK_SIZE * by + BLOCK_SIZE * ( gridDim.x - 1) + cols * ty + BLOCK_SIZE-1];
//  }
// 
//  __syncthreads();
//  
// 
//
//  temp[ty][tx]      = J_cuda[index];
//
//  __syncthreads();
//
//   jc = temp[ty][tx];
//
//   if ( ty == 0 && tx == 0 ){ //nw
//	n  = north[ty][tx] - jc;
//    s  = temp[ty+1][tx] - jc;
//    w  = west[ty][tx]  - jc; 
//    e  = temp[ty][tx+1] - jc;
//   }	    
//   else if ( ty == 0 && tx == BLOCK_SIZE-1 ){ //ne
//	n  = north[ty][tx] - jc;
//    s  = temp[ty+1][tx] - jc;
//    w  = temp[ty][tx-1] - jc; 
//    e  = east[ty][tx] - jc;
//   }
//   else if ( ty == BLOCK_SIZE -1 && tx == BLOCK_SIZE - 1){ //se
//	n  = temp[ty-1][tx] - jc;
//    s  = south[ty][tx] - jc;
//    w  = temp[ty][tx-1] - jc; 
//    e  = east[ty][tx]  - jc;
//   }
//   else if ( ty == BLOCK_SIZE -1 && tx == 0 ){//sw
//	n  = temp[ty-1][tx] - jc;
//    s  = south[ty][tx] - jc;
//    w  = west[ty][tx]  - jc; 
//    e  = temp[ty][tx+1] - jc;
//   }
//
//   else if ( ty == 0 ){ //n
//	n  = north[ty][tx] - jc;
//    s  = temp[ty+1][tx] - jc;
//    w  = temp[ty][tx-1] - jc; 
//    e  = temp[ty][tx+1] - jc;
//   }
//   else if ( tx == BLOCK_SIZE -1 ){ //e
//	n  = temp[ty-1][tx] - jc;
//    s  = temp[ty+1][tx] - jc;
//    w  = temp[ty][tx-1] - jc; 
//    e  = east[ty][tx] - jc;
//   }
//   else if ( ty == BLOCK_SIZE -1){ //s
//	n  = temp[ty-1][tx] - jc;
//    s  = south[ty][tx] - jc;
//    w  = temp[ty][tx-1] - jc; 
//    e  = temp[ty][tx+1] - jc;
//   }
//   else if ( tx == 0 ){ //w
//	n  = temp[ty-1][tx] - jc;
//    s  = temp[ty+1][tx] - jc;
//    w  = west[ty][tx] - jc; 
//    e  = temp[ty][tx+1] - jc;
//   }
//   else{  //the data elements which are not on the borders 
//	n  = temp[ty-1][tx] - jc;
//    s  = temp[ty+1][tx] - jc;
//    w  = temp[ty][tx-1] - jc; 
//    e  = temp[ty][tx+1] - jc;
//   }
//
//
//    g2 = ( n * n + s * s + w * w + e * e ) / (jc * jc);
//
//    l = ( n + s + w + e ) / jc;
//
//	num  = (0.5*g2) - ((1.0/16.0)*(l*l)) ;
//	den  = 1 + (.25*l);
//	qsqr = num/(den*den);
//
//	// diffusion coefficent (equ 33)
//	den = (qsqr-q0sqr) / (q0sqr * (1+q0sqr)) ;
//	c = 1.0 / (1.0+den) ;
//
//    // saturate diffusion coefficent
//	if (c < 0){temp_result[ty][tx] = 0;}
//	else if (c > 1) {temp_result[ty][tx] = 1;}
//	else {temp_result[ty][tx] = c;}
//
//    __syncthreads();
//
//    C_cuda[index] = temp_result[ty][tx];
//	E_C[index] = e;
//	W_C[index] = w;
//	S_C[index] = s;
//	N_C[index] = n;
//
//}
//
//__global__ void
//srad_cuda_2(
//		  float *E_C, 
//		  float *W_C, 
//		  float *N_C, 
//		  float *S_C,	
//		  float * J_cuda, 
//		  float * C_cuda, 
//		  int cols, 
//		  int rows, 
//		  float lambda,
//		  float q0sqr
//) 
//{
//	//block id
//	int bx = blockIdx.x;
//    int by = blockIdx.y;
//
//	//thread id
//    int tx = threadIdx.x;
//    int ty = threadIdx.y;
//
//	//indices
//    int index   = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + tx;
//	int index_s = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * BLOCK_SIZE + tx;
//    int index_e = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + BLOCK_SIZE;
//	float cc, cn, cs, ce, cw, d_sum;
//
//	//shared memory allocation
//	__shared__ float south_c[BLOCK_SIZE][BLOCK_SIZE];
//    __shared__ float  east_c[BLOCK_SIZE][BLOCK_SIZE];
//
//    __shared__ float c_cuda_temp[BLOCK_SIZE][BLOCK_SIZE];
//    __shared__ float c_cuda_result[BLOCK_SIZE][BLOCK_SIZE];
//    __shared__ float temp[BLOCK_SIZE][BLOCK_SIZE];
//
//    //load data to shared memory
//	temp[ty][tx]      = J_cuda[index];
//
//    __syncthreads();
//	 
//	south_c[ty][tx] = C_cuda[index_s];
//
//	if ( by == gridDim.y - 1 ){
//	south_c[ty][tx] = C_cuda[cols * BLOCK_SIZE * (gridDim.y - 1) + BLOCK_SIZE * bx + cols * ( BLOCK_SIZE - 1 ) + tx];
//	}
//	__syncthreads();
//	 
//	 
//	east_c[ty][tx] = C_cuda[index_e];
//	
//	if ( bx == gridDim.x - 1 ){
//	east_c[ty][tx] = C_cuda[cols * BLOCK_SIZE * by + BLOCK_SIZE * ( gridDim.x - 1) + cols * ty + BLOCK_SIZE-1];
//	}
//	 
//    __syncthreads();
//  
//    c_cuda_temp[ty][tx]      = C_cuda[index];
//
//    __syncthreads();
//
//	cc = c_cuda_temp[ty][tx];
//
//   if ( ty == BLOCK_SIZE -1 && tx == BLOCK_SIZE - 1){ //se
//	cn  = cc;
//    cs  = south_c[ty][tx];
//    cw  = cc; 
//    ce  = east_c[ty][tx];
//   } 
//   else if ( tx == BLOCK_SIZE -1 ){ //e
//	cn  = cc;
//    cs  = c_cuda_temp[ty+1][tx];
//    cw  = cc; 
//    ce  = east_c[ty][tx];
//   }
//   else if ( ty == BLOCK_SIZE -1){ //s
//	cn  = cc;
//    cs  = south_c[ty][tx];
//    cw  = cc; 
//    ce  = c_cuda_temp[ty][tx+1];
//   }
//   else{ //the data elements which are not on the borders 
//	cn  = cc;
//    cs  = c_cuda_temp[ty+1][tx];
//    cw  = cc; 
//    ce  = c_cuda_temp[ty][tx+1];
//   }
//
//   // divergence (equ 58)
//   d_sum = cn * N_C[index] + cs * S_C[index] + cw * W_C[index] + ce * E_C[index];
//
//   // image update (equ 61)
//   c_cuda_result[ty][tx] = temp[ty][tx] + 0.25 * lambda * d_sum;
//
//   __syncthreads();
//              
//   J_cuda[index] = c_cuda_result[ty][tx];
//    
//}
