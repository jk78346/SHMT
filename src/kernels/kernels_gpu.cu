#include "hip/hip_runtime.h"
#include <math.h>
#include <string>
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <assert.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/cudaarithm.hpp> // addWeighted()
#include <opencv2/cudafilters.hpp> // create[XXX]Filter()
#include "srad.h"
#include "BmpUtil.h"
#include "cuda_utils.h"
#include "kernels_gpu.h"
#include "srad_kernel.cu"
#include "kernels_fft.cuh"
#include "kernels_fft_wrapper.cu"
#include "dct8x8_kernel2.cuh"
#include "dct8x8_kernel_quantization.cuh"

#ifdef RD_WG_SIZE_0_0                                                            
        #define HOTSPOT_BLOCK_SIZE RD_WG_SIZE_0_0                                        
#elif defined(RD_WG_SIZE_0)                                                      
        #define HOTSPOT_BLOCK_SIZE RD_WG_SIZE_0                                          
#elif defined(RD_WG_SIZE)                                                        
        #define HOTSPOT_BLOCK_SIZE RD_WG_SIZE                                            
#else
        #define HOTSPOT_BLOCK_SIZE 16                                                            
#endif

/* some constants */
#define chip_height 0.016
#define chip_width 0.016
#define t_chip 0.0005
#define PRECISION 0.001
#define SPEC_HEAT_SI 1.75e6
#define K_SI 100
#define FACTOR_CHIP 0.5
#define MAX_PD 3.0e6

#define IN_RANGE(x, min, max)   ((x)>=(min) && (x)<=(max))
#define CLAMP_RANGE(x, min, max) x = (x<(min)) ? min : ((x>(max)) ? max : x )
//#define MIN(a, b) ((a)<=(b) ? (a) : (b))

#define BENCHMARK_SIZE 10
#define DCT_BLOCK_SIZE 8
#define DCT_BLOCK_SIZE2 64
#define DCT_BLOCK_SIZE_LOG2 3

/*
    GPU blackscholes
    Reference: samples/
*/
void GpuKernel::blackscholes_2d(KernelParams& kernel_params, void** in_img, void** out_img){
}

/*
    GPU dct8x8
    Reference: samples/3_Imaging/dct8x8/dct8x8.cu: CUDA2
*/
void GpuKernel::dct8x8_2d(KernelParams& kernel_params, void** in_img, void** out_img){
    /* integration code */
    float* ImgF1   = reinterpret_cast<float*>(*in_img);
    float* out_tmp = reinterpret_cast<float*>(*out_img);
    // a hard-coded params that used by this kernel.
    //int ImgStride;

    //allocate device memory
    float *src, *dst;
    ROI Size;
    Size.width  = kernel_params.params.get_kernel_size();
    Size.height = kernel_params.params.get_kernel_size();
    /* integration code */
    int StrideF = (((int)ceil((Size.width*sizeof(float))/16.0f))*16) / sizeof(float);
 //   byte *ImgDst = MallocPlaneByte(Size.width, Size.height, &ImgStride);
    size_t DeviceStride;
    checkCudaErrors(hipMallocPitch((void **)&src, &DeviceStride, Size.width * sizeof(float), Size.height));
    checkCudaErrors(hipMallocPitch((void **)&dst, &DeviceStride, Size.width * sizeof(float), Size.height));
    DeviceStride /= sizeof(float);

    //copy from host memory to device
    checkCudaErrors(hipMemcpy2D(src, DeviceStride * sizeof(float),
                                 ImgF1, StrideF * sizeof(float),
                                 Size.width * sizeof(float), Size.height,
                                 hipMemcpyHostToDevice));

    dim3 GridFullWarps(Size.width / KER2_BLOCK_WIDTH, Size.height / KER2_BLOCK_HEIGHT, 1);
    dim3 ThreadsFullWarps(8, KER2_BLOCK_WIDTH/8, KER2_BLOCK_HEIGHT/8);

    //perform block-wise DCT processing and benchmarking
    const int numIterations = 100;

    for (int i = -1; i < numIterations; i++)
    {
        if (i == 0)
        {
            checkCudaErrors(hipDeviceSynchronize());
        }
 
        CUDAkernel2DCT<<<GridFullWarps, ThreadsFullWarps>>>(dst, src, (int)DeviceStride);
        getLastCudaError("Kernel execution failed");
    }
 
    checkCudaErrors(hipDeviceSynchronize());

    //setup execution parameters for quantization
    dim3 ThreadsSmallBlocks(DCT_BLOCK_SIZE, DCT_BLOCK_SIZE);
    dim3 GridSmallBlocks(Size.width / DCT_BLOCK_SIZE, Size.height / DCT_BLOCK_SIZE);

    // execute Quantization kernel
    CUDAkernelQuantizationFloat<<< GridSmallBlocks, ThreadsSmallBlocks >>>(dst, (int) DeviceStride);
    getLastCudaError("Kernel execution failed");
 
    //perform block-wise IDCT processing
    CUDAkernel2IDCT<<<GridFullWarps, ThreadsFullWarps >>>(src, dst, (int)DeviceStride);
    checkCudaErrors(hipDeviceSynchronize());
    getLastCudaError("Kernel execution failed");
    
    //copy quantized image block to host
    checkCudaErrors(hipMemcpy2D(out_tmp, StrideF *sizeof(float),
                                 src, DeviceStride *sizeof(float),
                                 Size.width *sizeof(float), Size.height,
                                 hipMemcpyDeviceToHost));
 
    //convert image back to byte representation
//    AddFloatPlane(128.0f, out_tmp, StrideF, Size);
//    CopyFloat2Byte(out_tmp, StrideF, ImgDst, ImgStride, Size);

    //clean up memory
    checkCudaErrors(hipFree(dst));
    checkCudaErrors(hipFree(src));
}

void GpuKernel::fft_2d_input_conversion(){
    this->input_array_type.device_fp  = this->input_array_type.host_fp;
}

void GpuKernel::fft_2d_output_conversion(){
    Mat result;
    const int kernelH = 7;
    const int kernelW = 6;
    const int   dataH = kernel_params.params.get_kernel_size();
    const int   dataW = kernel_params.params.get_kernel_size();
    const int    fftH = snapTransformSize(dataH + kernelH - 1); 
    const int    fftW = snapTransformSize(dataW + kernelW - 1); 

    assert(this->output_array_type.device_fp != NULL);

    array2mat(result, this->output_array_type.device_fp, fftH, fftW);
    Mat cropped = result(Range(0, dataH), Range(0, dataW)); 
    mat2array(cropped, this->output_array_type.host_fp);
    free(this->output_array_type.device_fp);
}    

/*
    GPU convolveFFT2D, this kernel used a fixed 7x6 convolving kernel.
    Reference: samples/3_Imaging/convolutionFFT2D/convolutionFFT2D.cu
*/
void GpuKernel::fft_2d(KernelParams& kernel_params, void** in_array, void** out_array){
    float* h_Data      = reinterpret_cast<float*>(*in_array);
    float* h_ResultGPU = reinterpret_cast<float*>(*out_array);
    
    float* h_Kernel;
    
    float* d_Data ;
    float* d_PaddedData;
    float* d_Kernel;
    float* d_PaddedKernel;

    fComplex* d_DataSpectrum;
    fComplex* d_KernelSpectrum;

    hipfftHandle fftPlanFwd, fftPlanInv;

    const int kernelH = 7;
    const int kernelW = 6;
    const int kernelY = 3;
    const int kernelX = 4;
    const int   dataH = kernel_params.params.get_kernel_size();
    const int   dataW = kernel_params.params.get_kernel_size();
    const int    fftH = snapTransformSize(dataH + kernelH - 1); 
    const int    fftW = snapTransformSize(dataW + kernelW - 1); 

    //printf("...allocating memory\n");
    float fft_2d_kernel_array[7*6] = {
        13, 12, 13,  0,  1,  1,
        0,  7,  8,  2,  8,  0,
        5,  9,  1, 11, 11,  3,
        14, 14,  8, 11,  0,  3,
        6,  8, 14, 13,  0, 10,
        10, 11, 14,  1,  2,  0,
        5, 15,  7,  5,  1,  7
    };
    h_Kernel = fft_2d_kernel_array;

    checkCudaErrors(hipMalloc((void **)&d_Data, dataH * dataW * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_Kernel, kernelH * kernelW * sizeof(float)));
    
    checkCudaErrors(hipMalloc((void **)&d_PaddedData,   fftH * fftW * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_PaddedKernel, fftH * fftW * sizeof(float)));

    checkCudaErrors(hipMalloc((void **)&d_DataSpectrum,   fftH * (fftW / 2 + 1) * sizeof(fComplex)));
    checkCudaErrors(hipMalloc((void **)&d_KernelSpectrum, fftH * (fftW / 2 + 1) * sizeof(fComplex)));
    checkCudaErrors(hipMemset(d_KernelSpectrum, 0, fftH * (fftW / 2 + 1) * sizeof(fComplex)));

    //printf("...creating R2C & C2R FFT plans for %i x %i\n", fftH, fftW);
    checkCudaErrors(hipfftPlan2d(&fftPlanFwd, fftH, fftW, HIPFFT_R2C));
    checkCudaErrors(hipfftPlan2d(&fftPlanInv, fftH, fftW, HIPFFT_C2R));

    //printf("...uploading to gpu and padding convolution kernel and input data\n");
    checkCudaErrors(hipMemcpy(d_Kernel, 
                               h_Kernel, 
                               kernelH * kernelW * sizeof(float), 
                               hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_Data,   
                               h_Data,   
                               dataH   * dataW *   sizeof(float), 
                               hipMemcpyHostToDevice));
    checkCudaErrors(hipMemset(d_PaddedKernel, 0, fftH * fftW * sizeof(float)));
    checkCudaErrors(hipMemset(d_PaddedData,   0, fftH * fftW * sizeof(float)));

    padKernel(
        d_PaddedKernel,
        d_Kernel,
        fftH,
        fftW,
        kernelH,
        kernelW,
        kernelY,
        kernelX
    );

    padDataClampToBorder(
        d_PaddedData,
        d_Data,
        fftH,
        fftW,
        dataH,
        dataW,
        kernelH,
        kernelW,
        kernelY,
        kernelX
    );

    //printf("...transforming convolution kernel\n");
    checkCudaErrors(hipfftExecR2C(fftPlanFwd, 
                 (hipfftReal *)d_PaddedKernel, 
                 (hipfftComplex *)d_KernelSpectrum));
    
    //printf("...running GPU FFT convolution\n");
    checkCudaErrors(hipDeviceSynchronize());
    
    checkCudaErrors(hipfftExecR2C(fftPlanFwd, 
                                 (hipfftReal *)d_PaddedData, 
                                 (hipfftComplex *)d_DataSpectrum));
    modulateAndNormalize(d_DataSpectrum, d_KernelSpectrum, fftH, fftW, 1);
    checkCudaErrors(hipfftExecC2R(fftPlanInv, 
                                 (hipfftComplex *)d_DataSpectrum, 
                                 (hipfftReal *)d_PaddedData));
 
    checkCudaErrors(hipDeviceSynchronize());
    
    float* tmp = (float *)malloc(fftH    * fftW * sizeof(float));;
    
    //printf("...reading back GPU convolution results\n");
    checkCudaErrors(hipMemcpy(tmp, d_PaddedData, fftH * fftW * sizeof(float), hipMemcpyDeviceToHost)); 
    h_ResultGPU = tmp;
    *out_array = (void*)h_ResultGPU;

    checkCudaErrors(hipfftDestroy(fftPlanInv));
    checkCudaErrors(hipfftDestroy(fftPlanFwd));

    checkCudaErrors(hipFree(d_DataSpectrum));
    checkCudaErrors(hipFree(d_KernelSpectrum));
    checkCudaErrors(hipFree(d_PaddedData));
    checkCudaErrors(hipFree(d_PaddedKernel));
    checkCudaErrors(hipFree(d_Data));
    checkCudaErrors(hipFree(d_Kernel));
}
 
__global__ void calculate_temp(int iteration,  //number of iteration
                               float *power,   //power input
                               float *temp_src,    //temperature input/output
                               float *temp_dst,    //temperature input/output
                               int grid_cols,  //Col of grid
                               int grid_rows,  //Row of grid
                               int border_cols,  // border offset 
                               int border_rows,  // border offset
                               float Cap,      //Capacitance
                               float Rx,
                               float Ry,
                               float Rz,
                               float step,
                               float time_elapsed){

        __shared__ float temp_on_cuda[HOTSPOT_BLOCK_SIZE][HOTSPOT_BLOCK_SIZE];
        __shared__ float power_on_cuda[HOTSPOT_BLOCK_SIZE][HOTSPOT_BLOCK_SIZE];
        __shared__ float temp_t[HOTSPOT_BLOCK_SIZE][HOTSPOT_BLOCK_SIZE]; // saving temparary temperature result
 
    float amb_temp = 80.0;
        float step_div_Cap;
        float Rx_1,Ry_1,Rz_1;

    int bx = blockIdx.x;
        int by = blockIdx.y;
 
    int tx=threadIdx.x;
    int ty=threadIdx.y;
 
    step_div_Cap=step/Cap;

    Rx_1=1/Rx;
    Ry_1=1/Ry;
    Rz_1=1/Rz;
 
        // each block finally computes result for a small block
        // after N iterations. 
        // it is the non-overlapping small blocks that cover 
        // all the input data
 
        // calculate the small block size
    int small_block_rows = HOTSPOT_BLOCK_SIZE-iteration*2;//EXPAND_RATE
    int small_block_cols = HOTSPOT_BLOCK_SIZE-iteration*2;//EXPAND_RATE

        // calculate the boundary for the block according to 
        // the boundary of its small block
        int blkY = small_block_rows*by-border_rows;
        int blkX = small_block_cols*bx-border_cols;
        int blkYmax = blkY+HOTSPOT_BLOCK_SIZE-1;
        int blkXmax = blkX+HOTSPOT_BLOCK_SIZE-1;

        // calculate the global thread coordination
    int yidx = blkY+ty;
    int xidx = blkX+tx;
 
        // load data if it is within the valid input range
    int loadYidx=yidx, loadXidx=xidx;
        int index = grid_cols*loadYidx+loadXidx;
 
    if(IN_RANGE(loadYidx, 0, grid_rows-1) && IN_RANGE(loadXidx, 0, grid_cols-1)){
            temp_on_cuda[ty][tx] = temp_src[index];  // Load the temperature data from global memory to shared memory
            power_on_cuda[ty][tx] = power[index];// Load the power data from global memory to shared memory
    }
    __syncthreads();

        // effective range within this block that falls within 
        // the valid range of the input data
        // used to rule out computation outside the boundary.
        int validYmin = (blkY < 0) ? -blkY : 0;
        int validYmax = (blkYmax > grid_rows-1) ? HOTSPOT_BLOCK_SIZE-1-(blkYmax-grid_rows+1) : HOTSPOT_BLOCK_SIZE-1;
        int validXmin = (blkX < 0) ? -blkX : 0;
        int validXmax = (blkXmax > grid_cols-1) ? HOTSPOT_BLOCK_SIZE-1-(blkXmax-grid_cols+1) : HOTSPOT_BLOCK_SIZE-1;
 
        int N = ty-1;
        int S = ty+1;
        int W = tx-1;
        int E = tx+1;
 
        N = (N < validYmin) ? validYmin : N;
        S = (S > validYmax) ? validYmax : S;
        W = (W < validXmin) ? validXmin : W;
        E = (E > validXmax) ? validXmax : E;
 
        bool computed;
        for (int i=0; i<iteration ; i++){
            computed = false;
            if( IN_RANGE(tx, i+1, HOTSPOT_BLOCK_SIZE-i-2) &&  \
                  IN_RANGE(ty, i+1, HOTSPOT_BLOCK_SIZE-i-2) &&  \
                  IN_RANGE(tx, validXmin, validXmax) && \
                  IN_RANGE(ty, validYmin, validYmax) ) {
                  computed = true;
                  temp_t[ty][tx] =   temp_on_cuda[ty][tx] + step_div_Cap * (power_on_cuda[ty][tx] +
                     (temp_on_cuda[S][tx] + temp_on_cuda[N][tx] - 2.0*temp_on_cuda[ty][tx]) * Ry_1 +
                     (temp_on_cuda[ty][E] + temp_on_cuda[ty][W] - 2.0*temp_on_cuda[ty][tx]) * Rx_1 +
                     (amb_temp - temp_on_cuda[ty][tx]) * Rz_1);
 
            }
            __syncthreads();
            if(i==iteration-1)
                break;
            if(computed)     //Assign the computation range
                temp_on_cuda[ty][tx]= temp_t[ty][tx];
            __syncthreads();
          }
 
      // update the global memory
      // after the last iteration, only threads coordinated within the 
      // small block perform the calculation and switch on ``computed''
      if (computed){
          temp_dst[index]= temp_t[ty][tx];
      }
}
/*
   compute N time steps
*/
 
int compute_tran_temp(float *MatrixPower,float *MatrixTemp[2], int col, int row, \
        int total_iterations, int num_iterations, int blockCols, int blockRows, int borderCols, int borderRows)
{
        dim3 dimBlock(HOTSPOT_BLOCK_SIZE, HOTSPOT_BLOCK_SIZE);
        dim3 dimGrid(blockCols, blockRows);
     
    float grid_height = chip_height / row;
    float grid_width = chip_width / col;
     
    float Cap = FACTOR_CHIP * SPEC_HEAT_SI * t_chip * grid_width * grid_height;
    float Rx = grid_width / (2.0 * K_SI * t_chip * grid_height);
    float Ry = grid_height / (2.0 * K_SI * t_chip * grid_width);
    float Rz = t_chip / (K_SI * grid_height * grid_width);
     
    float max_slope = MAX_PD / (FACTOR_CHIP * t_chip * SPEC_HEAT_SI);
    float step = PRECISION / max_slope;
    float t;
        float time_elapsed;
    time_elapsed=0.001;
 
        int src = 1, dst = 0;
 
    for (t = 0; t < total_iterations; t+=num_iterations) {
            int temp = src;
            src = dst;
            dst = temp;
            calculate_temp<<<dimGrid, dimBlock>>>(MIN(num_iterations, total_iterations-t), MatrixPower,MatrixTemp[src],MatrixTemp[dst],\
        col,row,borderCols, borderRows, Cap,Rx,Ry,Rz,step,time_elapsed);
    }
        return dst;
}

/* Reference code: rodinia_3.1/cuda/hotspot/hotspot.cu */
void GpuKernel::hotspot_2d(KernelParams& kernel_params, void** input, void** output){

    int dim = kernel_params.params.get_kernel_size();
    int grid_rows = dim;
    int grid_cols = dim;
    int size = dim * dim;

    /* some constants */
    int total_iterations = 1;
    int pyramid_height = 1;

    /* pyramid parameters */
    # define EXPAND_RATE 2// add one iteration will extend the pyramid base by 2 per each borderline
    int borderCols = (pyramid_height)*EXPAND_RATE/2;
    int borderRows = (pyramid_height)*EXPAND_RATE/2;
    int smallBlockCol = HOTSPOT_BLOCK_SIZE-(pyramid_height)*EXPAND_RATE;
    int smallBlockRow = HOTSPOT_BLOCK_SIZE-(pyramid_height)*EXPAND_RATE;
    int blockCols = grid_cols/smallBlockCol+((grid_cols%smallBlockCol==0)?0:1);
    int blockRows = grid_rows/smallBlockRow+((grid_rows%smallBlockRow==0)?0:1);

    /* host pointers */
    float* host_float_ptr = reinterpret_cast<float*>(*input);
    float* FilesavingTemp = host_float_ptr;
    float* FilesavingPower = &host_float_ptr[size];
    float* MatrixOut = reinterpret_cast<float*>(*output);

    /* device pointers */
    float *MatrixTemp[2], *MatrixPower;
    hipMalloc((void**)&MatrixTemp[0], sizeof(float)*size);
    hipMalloc((void**)&MatrixTemp[1], sizeof(float)*size);
    hipMemcpy(MatrixTemp[0], FilesavingTemp, sizeof(float)*size, hipMemcpyHostToDevice);
 
    hipMalloc((void**)&MatrixPower, sizeof(float)*size);
    hipMemcpy(MatrixPower, FilesavingPower, sizeof(float)*size, hipMemcpyHostToDevice);
    printf("Start computing the transient temperature\n");
    int ret = compute_tran_temp(MatrixPower,MatrixTemp,grid_cols,grid_rows, \
     total_iterations,pyramid_height, blockCols, blockRows, borderCols, borderRows);
    printf("Ending simulation\n");
    hipMemcpy(MatrixOut, MatrixTemp[ret], sizeof(float)*size, hipMemcpyDeviceToHost);
 
    //writeoutput(MatrixOut,grid_rows, grid_cols, ofile);
 
    hipFree(MatrixPower);
    hipFree(MatrixTemp[0]);
    hipFree(MatrixTemp[1]);
}

void GpuKernel::kmeans_2d(const cuda::GpuMat in_img, cuda::GpuMat& out_img){
}

void GpuKernel::laplacian_2d(const cuda::GpuMat in_img, cuda::GpuMat& out_img){
    auto laplacian = cuda::createLaplacianFilter(in_img.type(), in_img.type(), 3/*kernel size*/, 1/*scale*/, BORDER_DEFAULT);
    laplacian->apply(in_img, out_img);
    cuda::abs(out_img, out_img);
}

void GpuKernel::mean_2d(const cuda::GpuMat in_img, cuda::GpuMat& out_img){
    auto median = cuda::createBoxFilter(in_img.type(), in_img.type(), Size(3, 3),     Point(-1, -1), BORDER_DEFAULT);
    median->apply(in_img, out_img);
}

void GpuKernel::minimum_2d(const cuda::GpuMat in_img, cuda::GpuMat& out_img){
    out_img = in_img;
}

void GpuKernel::sobel_2d(const cuda::GpuMat in_img, cuda::GpuMat& out_img){

    cuda::GpuMat grad_x, grad_y;
    cuda::GpuMat abs_grad_x, abs_grad_y;

    int ddepth = CV_32F;
    auto sobel_dx = cuda::createSobelFilter(in_img.type(), ddepth, 1, 0, 3);
    auto sobel_dy = cuda::createSobelFilter(in_img.type(), ddepth, 0, 1, 3);
 
    sobel_dx->apply(in_img, grad_x);
    sobel_dy->apply(in_img, grad_y);
 
    cuda::abs(grad_x, abs_grad_x);
    cuda::abs(grad_y, abs_grad_y);
  
    cuda::addWeighted(abs_grad_x, 0.5, abs_grad_y, 0.5, 0, out_img);
}

void GpuKernel::srad_2d(KernelParams& kernel_params, void** input, void** output){
    int rows = kernel_params.params.get_kernel_size();
    int cols = kernel_params.params.get_kernel_size();
    int size_I, size_R, niter = 1, iter;
    float *I, *J, lambda=0.5, q0sqr, sum, sum2, tmp, meanROI, varROI;

    float *J_cuda;
    float *C_cuda;
    float *E_C, *W_C, *N_C, *S_C;

    unsigned int r1 = 0, r2 = rows-1, c1 = 0, c2 = cols-1; // need init
    float *c;

    size_I = cols * rows;
    size_R = (r1-r1+1)*(c2-c1+1);
 
    I = (float*)*input;
    J = (float*)*output;
    c = (float *)malloc(sizeof(float)* size_I);

    //Allocate device memory
    hipMalloc((void**)& J_cuda, sizeof(float)* size_I);
    hipMalloc((void**)& C_cuda, sizeof(float)* size_I);
    hipMalloc((void**)& E_C, sizeof(float)* size_I);
    hipMalloc((void**)& W_C, sizeof(float)* size_I);
    hipMalloc((void**)& S_C, sizeof(float)* size_I);
    hipMalloc((void**)& N_C, sizeof(float)* size_I);

    for (int k = 0;  k < size_I; k++ ) {
        J[k] = (float)exp(I[k]) ;
    }

    for(iter=0; iter < niter ; iter++){
        sum=0; sum2=0;
        for (int i=r1; i<=r2; i++) {
            for (int j=c1; j<=c2; j++) {
                tmp   = J[i * cols + j];
                sum  += tmp ;
                sum2 += tmp*tmp;
            }
        }
        meanROI = sum / size_R;
        varROI  = (sum2 / size_R) - meanROI*meanROI;
        q0sqr   = varROI / (meanROI*meanROI);

        //Currently the input size must be divided by 16 - the block size
        int block_x = cols/SRAD_BLOCK_SIZE ;
        int block_y = rows/SRAD_BLOCK_SIZE ;
 
        dim3 dimBlock(SRAD_BLOCK_SIZE, SRAD_BLOCK_SIZE);
        dim3 dimGrid(block_x , block_y);
 
        //Copy data from main memory to device memory
        hipMemcpy(J_cuda, J, sizeof(float) * size_I, hipMemcpyHostToDevice);
 
        //Run kernels
        srad_cuda_1<<<dimGrid, dimBlock>>>(E_C, W_C, N_C, S_C, J_cuda, C_cuda,     cols, rows, q0sqr);
        srad_cuda_2<<<dimGrid, dimBlock>>>(E_C, W_C, N_C, S_C, J_cuda, C_cuda,     cols, rows, lambda, q0sqr);
 
        //Copy data from device memory to main memory
        hipMemcpy(J, J_cuda, sizeof(float) * size_I, hipMemcpyDeviceToHost);
    }
    hipDeviceSynchronize();
    //hipDeviceSynchronize();

    hipFree(C_cuda);
    hipFree(J_cuda);
    hipFree(E_C);
    hipFree(W_C);
    hipFree(N_C);
    hipFree(S_C);
    free(c);
}

