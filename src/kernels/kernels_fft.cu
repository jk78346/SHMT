#include <string>
#include <stdio.h>
#include "cuda_utils.h"
#include "kernels_cpu.h"
#include "kernels_gpu.h"
#include "kernels_fft.cuh"

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
//#include <hip/hip_runtime_api.h>
//#include <hip/hip_runtime.h>

float fft_2d_kernel_array[7*6] = {
    13, 12, 13,  0,  1,  1,
     0,  7,  8,  2,  8,  0,
     5,  9,  1, 11, 11,  3,
    14, 14,  8, 11,  0,  3,
     6,  8, 14, 13,  0, 10,
    10, 11, 14,  1,  2,  0,
     5, 15,  7,  5,  1,  7
};
/*
    CPU convolveFFT2D, this kernel used a fixed 7x6 convolving kernel.
    Reference: samples/3_Imaging/convolutionFFT2D/convolutionFFT2D_gold.cpp
*/
void CpuKernel::fft_2d(Params params, float* input, float* output){
    float *h_Result = output;
    float *h_Data = input;
    float *h_Kernel = fft_2d_kernel_array;
    int dataH = params.get_kernel_size();
    int dataW = params.get_kernel_size();
    int kernelH = 7;
    int kernelW = 6;
    int kernelY = 3;
    int kernelX = 4;

    for (int y = 0; y < dataH; y++)
        for (int x = 0; x < dataW; x++)
        {
            double sum = 0;

            for (int ky = -(kernelH - kernelY - 1); ky <= kernelY; ky++)
                for (int kx = -(kernelW - kernelX - 1); kx <= kernelX; kx++)
                {
                    int dy = y + ky;
                    int dx = x + kx;

                    if (dy < 0) dy = 0;

                    if (dx < 0) dx = 0;

                    if (dy >= dataH) dy = dataH - 1;
 
                    if (dx >= dataW) dx = dataW - 1;
                    
                    sum += h_Data[dy * dataW + dx] * h_Kernel[(kernelY - ky) * kernelW + (kernelX - kx)];
                }

            h_Result[y * dataW + x] = (float)sum;
        }
}

void GpuKernel::fft_2d_input_conversion(Params params, float* input_array){
// ***** start to integrating fft_2d as the first integration trial *****
    const int kernelH = 7;
    const int kernelW = 6;
//            const int kernelY = 3;
//            const int kernelX = 4;
    const int   dataH = params.get_kernel_size();
    const int   dataW = params.get_kernel_size();
 
    const int fftH = snapTransformSize(dataH + kernelH - 1);
    const int fftW = snapTransformSize(dataW + kernelW - 1);
 
    float* h_Data = input_array;
    // Need to fill in the pre-determined kernel matrix for fair comparision
    float* h_Kernel = (float *)malloc(kernelH * kernelW * sizeof(float));
    float* h_ResultGPU = (float *)malloc(fftH    * fftW * sizeof(float));
    float *d_Data;
    float *d_Kernel;
    float* d_PaddedData;
    float* d_PaddedKernel;
 
    fComplex
    *d_DataSpectrum,
    *d_KernelSpectrum;
 
    hipfftHandle
    fftPlanFwd,
    fftPlanInv;
 
    hipMalloc((void **)&d_Data, dataH * dataW * sizeof(float));
    hipMalloc((void **)&d_Kernel, kernelH * kernelW * sizeof(float));
    hipMemcpy(d_Kernel, h_Kernel, kernelH * kernelW * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Data,   h_Data,   dataH   * dataW *   sizeof(float), hipMemcpyHostToDevice);
 
    hipMalloc((void **)&d_PaddedData,   fftH * fftW * sizeof(float));
    hipMalloc((void **)&d_PaddedKernel, fftH * fftW * sizeof(float));
 
    hipMalloc((void **)&d_DataSpectrum,   fftH * (fftW / 2 + 1) * sizeof(fComplex));
    hipMalloc((void **)&d_KernelSpectrum, fftH * (fftW / 2 + 1) * sizeof(fComplex));
    hipMemset(d_KernelSpectrum, 0, fftH * (fftW / 2 + 1) * sizeof(fComplex));
 
    printf("...creating R2C & C2R FFT plans for %i x %i\n", fftH, fftW);
    hipfftPlan2d(&fftPlanFwd, fftH, fftW, HIPFFT_R2C);
    hipfftPlan2d(&fftPlanInv, fftH, fftW, HIPFFT_C2R);

//    padKernel(
//        d_PaddedKernel,
//        d_Kernel,
//        fftH,
//        fftW,
//        kernelH,
//        kernelW,
//        kernelY,
//        kernelX
//    );

//    fft_2d_input_conversion_wrapper();
}

/*
    GPU convolveFFT2D, this kernel used a fixed 7x6 convolving kernel.
    Reference: samples/3_Imaging/convolutionFFT2D/convolutionFFT2D.cu
*/
void GpuKernel::fft_2d(Params params, float* in_img, float* out_img){
    fft_2d_kernel_wrapper(in_img, out_img);
//    //Not including kernel transformation into time measurement,
//    //since convolution kernel is not changed very frequently
//    printf("...transforming convolution kernel\n");
//    timing kernel_fft_s = clk::now();
//    checkCudaErrors(hipfftExecR2C(fftPlanFwd, (hipfftReal *)d_PaddedKernel, (cuff    tComplex *)d_KernelSpectrum));
//    timing kernel_fft_e = clk::now();
// 
//    printf("...running GPU FFT convolution: ");
//    checkCudaErrors(hipDeviceSynchronize());
//    sdkResetTimer(&hTimer);
//    sdkStartTimer(&hTimer);
//
//    for(int i = 0 ; i < iter ; i++){
//        checkCudaErrors(hipfftExecR2C(fftPlanFwd, (hipfftReal *)d_PaddedData, (cu    fftComplex *)d_DataSpectrum));
//        modulateAndNormalize(d_DataSpectrum, d_KernelSpectrum, fftH, fftW, 1);
//        checkCudaErrors(hipfftExecC2R(fftPlanInv, (hipfftComplex *)d_DataSpectrum    , (hipfftReal *)d_PaddedData));
// 
//        checkCudaErrors(hipDeviceSynchronize());
//    }
//    sdkStopTimer(&hTimer);
//    double gpuTime = sdkGetTimerValue(&hTimer)/iter;
//    printf("%f MPix/s (%f ms), averaged over %d time(s)\n", (double)dataH * (do    uble)dataW * 1e-6 / (gpuTime * 0.001), gpuTime, iter);
// 
//    printf("...reading back GPU convolution results\n");
//    checkCudaErrors(hipMemcpy(h_ResultGPU, d_PaddedData, fftH * fftW * sizeof(    float), hipMemcpyDeviceToHost));
}


