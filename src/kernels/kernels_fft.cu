#include <string>
#include <stdio.h>
#include "cuda_utils.h"
#include "kernels_cpu.h"
#include "kernels_gpu.h"
#include "kernels_fft.cuh"
#include "kernels_fft_wrapper.cu"

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
//#include <hip/hip_runtime_api.h>
//#include <hip/hip_runtime.h>

float fft_2d_kernel_array[7*6] = {
    13, 12, 13,  0,  1,  1,
     0,  7,  8,  2,  8,  0,
     5,  9,  1, 11, 11,  3,
    14, 14,  8, 11,  0,  3,
     6,  8, 14, 13,  0, 10,
    10, 11, 14,  1,  2,  0,
     5, 15,  7,  5,  1,  7
};
/*
    CPU convolveFFT2D, this kernel used a fixed 7x6 convolving kernel.
    Reference: samples/3_Imaging/convolutionFFT2D/convolutionFFT2D_gold.cpp
*/
void CpuKernel::fft_2d(Params params, float* input, float* output){
    float *h_Result = output;
    float *h_Data = input;
    float *h_Kernel = fft_2d_kernel_array;
    int dataH = params.get_kernel_size();
    int dataW = params.get_kernel_size();
    int kernelH = 7;
    int kernelW = 6;
    int kernelY = 3;
    int kernelX = 4;

    for (int y = 0; y < dataH; y++)
        for (int x = 0; x < dataW; x++)
        {
            double sum = 0;

            for (int ky = -(kernelH - kernelY - 1); ky <= kernelY; ky++)
                for (int kx = -(kernelW - kernelX - 1); kx <= kernelX; kx++)
                {
                    int dy = y + ky;
                    int dx = x + kx;

                    if (dy < 0) dy = 0;

                    if (dx < 0) dx = 0;

                    if (dy >= dataH) dy = dataH - 1;
 
                    if (dx >= dataW) dx = dataW - 1;
                    
                    sum += h_Data[dy * dataW + dx] * h_Kernel[(kernelY - ky) * kernelW + (kernelX - kx)];
                }

            h_Result[y * dataW + x] = (float)sum;
        }
}

void GpuKernel::fft_2d_input_conversion(){
    this->input_array_type.device_fp  = this->input_array_type.host_fp;
}

void GpuKernel::fft_2d_output_conversion(){
    Mat result;
    const int kernelH = 7;
    const int kernelW = 6;
    const int   dataH = kernel_params.params.get_kernel_size();
    const int   dataW = kernel_params.params.get_kernel_size();
    const int    fftH = snapTransformSize(dataH + kernelH - 1); 
    const int    fftW = snapTransformSize(dataW + kernelW - 1); 

    assert(this->output_array_type.device_fp != NULL);

    array2mat(result, this->output_array_type.device_fp, fftH, fftW);
    Mat cropped = result(Range(0, dataH), Range(0, dataW)); 
    mat2array(cropped, this->output_array_type.host_fp);
}    

/*
    GPU convolveFFT2D, this kernel used a fixed 7x6 convolving kernel.
    Reference: samples/3_Imaging/convolutionFFT2D/convolutionFFT2D.cu
*/
void GpuKernel::fft_2d(KernelParams& kernel_params, void** in_array, void** out_array){
    float* h_Data      = reinterpret_cast<float*>(*in_array);
    float* h_ResultGPU = reinterpret_cast<float*>(*out_array);
    
    float* h_Kernel;
    
    float* d_Data ;
    float* d_PaddedData;
    float* d_Kernel;
    float* d_PaddedKernel;

    fComplex* d_DataSpectrum;
    fComplex* d_KernelSpectrum;

    hipfftHandle fftPlanFwd, fftPlanInv;

    const int kernelH = 7;
    const int kernelW = 6;
    const int kernelY = 3;
    const int kernelX = 4;
    const int   dataH = kernel_params.params.get_kernel_size();
    const int   dataW = kernel_params.params.get_kernel_size();
    const int    fftH = snapTransformSize(dataH + kernelH - 1); 
    const int    fftW = snapTransformSize(dataW + kernelW - 1); 

    //printf("...allocating memory\n");
    h_Kernel = fft_2d_kernel_array;

    checkCudaErrors(hipMalloc((void **)&d_Data, dataH * dataW * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_Kernel, kernelH * kernelW * sizeof(float)));
    
    checkCudaErrors(hipMalloc((void **)&d_PaddedData,   fftH * fftW * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_PaddedKernel, fftH * fftW * sizeof(float)));

    checkCudaErrors(hipMalloc((void **)&d_DataSpectrum,   fftH * (fftW / 2 + 1) * sizeof(fComplex)));
    checkCudaErrors(hipMalloc((void **)&d_KernelSpectrum, fftH * (fftW / 2 + 1) * sizeof(fComplex)));
    checkCudaErrors(hipMemset(d_KernelSpectrum, 0, fftH * (fftW / 2 + 1) * sizeof(fComplex)));

    //printf("...creating R2C & C2R FFT plans for %i x %i\n", fftH, fftW);
    checkCudaErrors(hipfftPlan2d(&fftPlanFwd, fftH, fftW, HIPFFT_R2C));
    checkCudaErrors(hipfftPlan2d(&fftPlanInv, fftH, fftW, HIPFFT_C2R));

    //printf("...uploading to gpu and padding convolution kernel and input data\n");
    checkCudaErrors(hipMemcpy(d_Kernel, 
                               h_Kernel, 
                               kernelH * kernelW * sizeof(float), 
                               hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_Data,   
                               h_Data,   
                               dataH   * dataW *   sizeof(float), 
                               hipMemcpyHostToDevice));
    checkCudaErrors(hipMemset(d_PaddedKernel, 0, fftH * fftW * sizeof(float)));
    checkCudaErrors(hipMemset(d_PaddedData,   0, fftH * fftW * sizeof(float)));

    padKernel(
        d_PaddedKernel,
        d_Kernel,
        fftH,
        fftW,
        kernelH,
        kernelW,
        kernelY,
        kernelX
    );

    padDataClampToBorder(
        d_PaddedData,
        d_Data,
        fftH,
        fftW,
        dataH,
        dataW,
        kernelH,
        kernelW,
        kernelY,
        kernelX
    );

    //printf("...transforming convolution kernel\n");
    checkCudaErrors(hipfftExecR2C(fftPlanFwd, 
                 (hipfftReal *)d_PaddedKernel, 
                 (hipfftComplex *)d_KernelSpectrum));
    
    //printf("...running GPU FFT convolution\n");
    checkCudaErrors(hipDeviceSynchronize());
    
    checkCudaErrors(hipfftExecR2C(fftPlanFwd, 
                                 (hipfftReal *)d_PaddedData, 
                                 (hipfftComplex *)d_DataSpectrum));
    modulateAndNormalize(d_DataSpectrum, d_KernelSpectrum, fftH, fftW, 1);
    checkCudaErrors(hipfftExecC2R(fftPlanInv, 
                                 (hipfftComplex *)d_DataSpectrum, 
                                 (hipfftReal *)d_PaddedData));
 
    checkCudaErrors(hipDeviceSynchronize());
    
    float* tmp = (float *)malloc(fftH    * fftW * sizeof(float));;
    
    //printf("...reading back GPU convolution results\n");
    checkCudaErrors(hipMemcpy(tmp, d_PaddedData, fftH * fftW * sizeof(float), hipMemcpyDeviceToHost)); 
    h_ResultGPU = tmp;
    *out_array = (void*)h_ResultGPU;

    checkCudaErrors(hipfftDestroy(fftPlanInv));
    checkCudaErrors(hipfftDestroy(fftPlanFwd));

    checkCudaErrors(hipFree(d_DataSpectrum));
    checkCudaErrors(hipFree(d_KernelSpectrum));
    checkCudaErrors(hipFree(d_PaddedData));
    checkCudaErrors(hipFree(d_PaddedKernel));
    checkCudaErrors(hipFree(d_Data));
    checkCudaErrors(hipFree(d_Kernel));
}

